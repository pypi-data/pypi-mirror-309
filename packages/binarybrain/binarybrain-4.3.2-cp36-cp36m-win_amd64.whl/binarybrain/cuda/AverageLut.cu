#include "hip/hip_runtime.h"
﻿#include <iostream>
#include <chrono>
#include <algorithm>

#include "hip/hip_runtime.h"
#include ""

#include "bbcu/bbcu.h"
#include "bbcu/bbcu_util.h"



// -------------------------------------------------
//  Forward
// -------------------------------------------------


template <typename T>
__global__ void kernal_AverageLut_Forward(
            T   const   *x_buf,
            T           *y_buf,
            int const   *input_index,
            int         n,
            T           reciprocal_n,
            int         node_size,
            int         frame_size,
            int         frame_stride,
            bool        binarize_input,
            bool        binarize_output
        )
{
    int frame = blockIdx.x * blockDim.x + threadIdx.x;
    int node  = blockIdx.y * blockDim.y + threadIdx.y;

    if ( node < node_size && frame < frame_size ) {
        T   sum = 0;
        for ( int i = 0; i < n; ++i ) {
            int input_node = input_index[node*n + i];
            T   x = x_buf[input_node*frame_stride + frame];
            if (binarize_input) {
                x = (x > 0) ? (T)BB_BINARY_HI : (T)BB_BINARY_LO;
            }
            sum += x;
        }

        T   average;
        if (binarize_output) {
            average = (sum > 0) ? (T)BB_BINARY_HI : (T)BB_BINARY_LO;
        }
        else {
            average = sum * reciprocal_n;
        }

        y_buf[node*frame_stride + frame] = average;
    }
}


template <typename T>
BBCU_DLL_EXPORT int bbcu_AverageLut_Forward
        (
            T   const       *dev_x_buf,
            T               *dev_y_buf,
            int const       *dev_input_index,
            int             n,
            int             node_size,
            int             frame_size,
            int             frame_stride,
            bool            binarize_input,
            bool            binarize_output,
            hipStream_t    streamId
        )
{
    BBCU_DEBUG_ASSERT(bbcu_IsDeviceAvailable());

    dim3    block(1024, 1);
    while (block.x / 2 >= (unsigned int)frame_size) {
        block.x /= 2;
        block.y *= 2;
    }
    block.x = std::min(block.x, (unsigned int)frame_size);
    block.y = std::min(block.y, (unsigned int)node_size);

    dim3    grid;
    grid.x = (frame_size + block.x - 1) / block.x;
    grid.y = (node_size  + block.y - 1) / block.y;
    
    kernal_AverageLut_Forward<T><<<grid, block, 0, streamId>>>(
            dev_x_buf,
            dev_y_buf,
            dev_input_index,
            n,
            (T)1.0 / (T)n,
            node_size,
            frame_size,
            frame_stride,
            binarize_input,
            binarize_output
        );
    BB_CUDA_CHECK_LAST_ERROR();
    
    return 0;
}

template BBCU_DLL_EXPORT int bbcu_AverageLut_Forward<float >(float  const *, float  *, int const *, int, int, int, int, bool, bool, hipStream_t);
template BBCU_DLL_EXPORT int bbcu_AverageLut_Forward<double>(double const *, double *, int const *, int, int, int, int, bool, bool, hipStream_t);



__global__ void kernal_bit_AverageLut_Forward(
            int const   *x_buf,
            int         *y_buf,
            int const   *input_index,
            int         n,
            int         node_size,
            int         frame_size,
            int         frame_stride
        )
{
    int frame = blockIdx.x * blockDim.x + threadIdx.x;
    int node  = blockIdx.y * blockDim.y + threadIdx.y;

    if ( node < node_size && frame < frame_size ) {
        int bitmask = 1;
        int y = 0;
        for ( int bitpos = 0; bitpos < 32; ++bitpos ) {
            int count = 0;
            for ( int i = 0; i < n; ++i ) {
                int input_node = input_index[node*n + i];
                int x = x_buf[input_node*frame_stride + frame];
                count += (x & bitmask) ? +1 : -1;
            }
            y |= (count > 0) ? bitmask : 0;
            bitmask <<= 1;
        }
        y_buf[node*frame_stride + frame] = y;
    }
}

BBCU_DLL_EXPORT int bbcu_bit_AverageLut_Forward
        (
            int const       *dev_x_buf,
            int             *dev_y_buf,
            int const       *dev_input_index,
            int             n,
            int             node_size,
            int             frame_size,
            int             frame_stride,
            hipStream_t    streamId
        )
{
    BBCU_DEBUG_ASSERT(bbcu_IsDeviceAvailable());

    frame_size = (frame_size + 31) / 32;

    dim3    block(1024, 1);
    while (block.x / 2 >= (unsigned int)frame_size) {
        block.x /= 2;
        block.y *= 2;
    }
    block.x = std::min(block.x, (unsigned int)frame_size);
    block.y = std::min(block.y, (unsigned int)node_size);

    dim3    grid;
    grid.x = (frame_size + block.x - 1) / block.x;
    grid.y = (node_size  + block.y - 1) / block.y;
    
    kernal_bit_AverageLut_Forward<<<grid, block, 0, streamId>>>(
            dev_x_buf,
            dev_y_buf,
            dev_input_index,
            n,
            node_size,
            frame_size,
            frame_stride
        );
    BB_CUDA_CHECK_LAST_ERROR();
    
    return 0;
}



// -------------------------------------------------
//  Backward
// -------------------------------------------------

template <typename T>
__global__ void kernal_AverageLut_Backward(
            T   const   *dy_buf,
            T           *dx_buf,
            int const   *input_index,
            int         n,
            T           reciprocal_n,
            int         node_size,
            int         frame_size,
            int         frame_stride
        )
{
    int frame = blockIdx.x * blockDim.x + threadIdx.x;

    if ( frame < frame_size ) {
        for ( int node = 0; node < node_size; ++node ) {
            T dy = dy_buf[node*frame_stride + frame] * reciprocal_n;
            for ( int i = 0; i < n; ++i ) {
                int input_node = input_index[node*n + i];
                dx_buf[input_node*frame_stride + frame] += dy;
            }
        }
    }
}


template <typename T>
BBCU_DLL_EXPORT int bbcu_AverageLut_Backward
        (
            T   const       *dev_dy_buf,
            T               *dev_dx_buf,
            int const       *dev_input_index,
            int             n,
            int             input_node_size,
            int             output_node_size,
            int             frame_size,
            int             frame_stride,
            hipStream_t    streamId
        )
{
    BBCU_DEBUG_ASSERT(bbcu_IsDeviceAvailable());

    hipMemset(dev_dx_buf, 0, sizeof(T)*frame_stride*input_node_size);

    dim3    block(std::min(frame_size, 1024));
    dim3    grid((frame_size + 1023) / 1024);
    
    kernal_AverageLut_Backward<T><<<grid, block, 0, streamId>>>(
            dev_dy_buf,
            dev_dx_buf,
            dev_input_index,
            n,
            (T)1/ (T)n,
            output_node_size,
            frame_size,
            frame_stride
        );
    BB_CUDA_CHECK_LAST_ERROR();
    
    return 0;
}

template BBCU_DLL_EXPORT int bbcu_AverageLut_Backward<float >(float  const *, float  *, int const *, int, int, int, int, int, hipStream_t);
template BBCU_DLL_EXPORT int bbcu_AverageLut_Backward<double>(double const *, double *, int const *, int, int, int, int, int, hipStream_t);



// end of file
